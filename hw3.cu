#include "hip/hip_runtime.h"
/*
KAM PUI SO (ANTHONY)
CS 510 GPU
Homework 3

1. Complete problems 8.8 and 8.10 in the text (p. 196)
2. Compare the performance of the two different 2D versions of code 
   (one uses constant memory and one does not) on 6 different 
   array sizes.
Submit your code and your performance results by email to: 
   karavan@pdx.edu with subject: 
   GPU HW3 You can submit files of type .cu, .txt, .pdf, .doc 
   You can include your performance results directly in the email 
   if that is easier. 
*/

#include <sys/time.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define WIDTH 10
#define HEIGHT 10
#define MASK_WIDTH 3
#define MASK_HEIGHT 3
#define MAX 2000
#define LIMIT 32
#define RANGE 10
#define ROW 0
#define COL 1

// global const

const int DIAGMASK[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
const int VERTMASK[9] = {1, 0, 0, 1, 0, 0, 1, 0, 0};


// The kernel that will execute on the GPU
__global__ void basic_2d_kernel(int *start, int *mask, int *result, int width, int height, int mask_width, int mask_height) {
    // declare kernel variable
    int center_x = blockDim.x * blockIdx.x + threadIdx.x;
    int center_y = blockDim.y * blockIdx.y + threadIdx.y;
    int current_x, current_y;
    int n_x_start_point = center_x - (mask_width / 2);
    int n_y_start_point = center_y - (mask_height / 2);
    int pvalue = 0;    

    // loop thru the mask area for one location
    for (int y = 0; y < mask_height; y++) {
        current_y = (n_y_start_point + y + height) % height;
        if ((current_y >= 0) && (current_y < height)) {
            for (int x = 0; x < mask_width; x++) {
                current_x = (n_x_start_point + x + width) % width;
                if ((current_x >= 0) && (current_x < width)) {
                    pvalue += start[(current_y * width) + current_x] * mask[(y * mask_width) + x];
                }
            }
        }
    }
    result[(center_y * width) + center_x] = pvalue;
}


// This function encapsulates the process of creating and tearing down the
// environment used to execute our game of life iteration kernel. The steps of the
// process are:
//   1. Allocate memory on the device to hold our board vectors
//   2. Copy the board vectors to device memory
//   3. Execute the kernel
//   4. Retrieve the result board vector from the device by copying it to the host
//   5. Free memory on the device
//
void basic_2d_dev(int *start, int *mask, int *result, int width, int height, int mask_width, int mask_height) {
    // Step 1: Allocate memory
    int *start_dev, *mask_dev,  *result_dev;
    int n = width * height;
    int m = mask_width * mask_height;

    // Since hipMalloc does not return a pointer like C's traditional malloc
    // (it returns a success status instead), we provide as it's first argument
    // the address of our device pointer variable so that it can change the
    // value of our pointer to the correct device address.
    hipMalloc((void **) &start_dev, sizeof(int) * n);
    hipMalloc((void **) &result_dev, sizeof(int) * n);
    hipMalloc((void **) &mask_dev, sizeof(int) * m);

    // Step 2: Copy the input vectors to the device
    hipMemcpy(start_dev, start, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(mask_dev, mask, sizeof(int) * m, hipMemcpyHostToDevice);

    // Step 3: Invoke the kernel
    dim3 dimGrid(LIMIT, LIMIT, 1);
    dim3 dimBlock(ceil(width/ (float) LIMIT), ceil(height/ (float) LIMIT), 1);
    basic_2d_kernel<<<dimGrid, dimBlock>>>(start_dev, mask_dev, result_dev, width, height, mask_width, mask_height);

    // Step 4: Retrieve the results
    hipMemcpy(result, result_dev, sizeof(int) * n, hipMemcpyDeviceToHost);

    // Step 5: Free device memory
    hipFree(start_dev);
    hipFree(mask_dev);
    hipFree(result_dev);
}



// The old-fashioned CPU-only way
void basic_2d_host(int *start, int *mask, int *result, int width, int height, int mask_width, int mask_height) {

    int x;
    int y;
    int m_x;
    int m_y;
    int n_x;
    int n_y;
    int offset_x = mask_width / 2;
    int offset_y = mask_height / 2;
    int pvalue = 0;

    for (y = 0; y < height; ++y) {
        for (x = 0; x < width; ++x) {
            pvalue = 0;
            for (m_y = 0; m_y < mask_height; m_y++) {
               for (m_x = 0; m_x < mask_width; m_x++) {
                  n_x = (x + m_x - (offset_x) + width) % width;
                  n_y = (y + m_y - (offset_y) + height) % height;
                  pvalue += (start[n_y * width + n_x] * mask[m_y * mask_width + m_x]);
               }
            }
            result[y * width + x] = pvalue;
        }
    } 
}



// fill the mask with random values
void fill_image(int *image, int width, int height, int scale) {
    int i;
    for (i = 0; i < (width * height); i++)
        image[i] = rand() % scale;
}


// fill the mask with pattern values
void fill_pattern(int *image, int width, int height, int scale) {
    int i;
    for (i = 0; i < (width * height); i++) {
        if (i % (width / 2))
            image[i] = 0;
        else
            image[i] = rand() % scale;
    }
}


// print divider
void print_divider() {
    printf("---------------------------------------\n");
}



// print image
void print_image(int *image, int width, int height) {
    int x, y;
    for (y = 0; y<height; y++) {
        for (x = 0; x<width; x++) {
            printf("%d ", image[y * width + x]);
        }
        printf("\n");
    }
    print_divider();
}


// normalize mask image
void normalize_image(int *image, int width, int height, int scale) {
    int i;
    int max = image[0];

    // find max and min
    for (i = 0; i < (width * height); i++) {
        if (image[i] > max)
            max = image[i];
    }

    for (i = 0; i < (width * height); i++) {
        image[i] = (int) ((float) image[i] / (float) max * (float) (scale -1)) ;
    }
}


// show device capability
void device_check() {
    int deviceCount;
    int device;

    hipGetDeviceCount(&deviceCount);
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        print_divider();
        printf("Device %d has compute capability %d.%d\n", 
               device, deviceProp.major, deviceProp.minor);
	printf("Max Threads per Block: %d \n", deviceProp.maxThreadsPerBlock);
	printf("Max Threads for x direction per Block: %d \n", deviceProp.maxThreadsDim[0]);
	printf("Max Threads for y direction per Block: %d \n", deviceProp.maxThreadsDim[1]);
	printf("Max Threads for z direction per Block: %d \n", deviceProp.maxThreadsDim[2]);
	printf("Max Blocks for x direction per Grid: %d \n", deviceProp.maxGridSize[0]);
	printf("Max Blocks for y direction per Grid: %d \n", deviceProp.maxGridSize[1]);
	printf("Max Blocks for z direction per Grid: %d \n", deviceProp.maxGridSize[2]);
        printf("Max Warp Size: %d \n", deviceProp.warpSize);
        printf("Number of SM: %d \n", deviceProp.multiProcessorCount);
        printf("Max Threads per SM: %d \n", deviceProp.maxThreadsPerMultiProcessor);
        printf("Number of Registers in each SM: %d \n", deviceProp.regsPerBlock);
        printf("Amount of Shared Memory Available: %zd \n", deviceProp.sharedMemPerBlock);
        printf("Amount of Constant Memory Available: %zd \n", deviceProp.totalConstMem);
        printf("Amount of Global Memory Available: %zd \n", deviceProp.totalGlobalMem);
        printf("Clock Rate: %d \n", deviceProp.clockRate);
        print_divider();
    }
}


// print different of two times
void print_time(timeval begin, timeval end) {
    printf("Time = %ld us\n", ((end.tv_sec * 1000000 + end.tv_usec) - (begin.tv_sec * 1000000 + begin.tv_usec )));
}


// main function
int main(void) {
    // image variable
    int n = WIDTH * HEIGHT;
    int *start = (int *) malloc(n* sizeof(int));
    int *result = (int *) malloc(n * sizeof(int));

    // mask variable 
    int m = MASK_WIDTH * MASK_HEIGHT;
//    int *mask = (int *) malloc(m * sizeof(int));
//    int *mask = (int *) VERTMASK;    // static mask with vertical 1's       
    int *mask = (int *) DIAGMASK;    // static mask with diagonal 1's

    // additional variable
    int i = 0;
    int *temp;

    // time variable
    struct timeval begin, end;

    // initialize rand seed
    srand(time(NULL));

    // check device property (warm up device...)
    device_check();

    // initialize the mask image and global image
    print_divider();
//    fill_image(mask, MASK_WIDTH, MASK_HEIGHT, RANGE);
    print_image(mask, MASK_WIDTH, MASK_HEIGHT);
    fill_image(start, WIDTH, HEIGHT, RANGE);
//    fill_pattern(start, WIDTH, HEIGHT, RANGE);
    print_image(start, WIDTH, HEIGHT);

    // run 2d convulotion with timer and print result
    gettimeofday(&begin, NULL);
//    basic_2d_host(start, mask, result, WIDTH, HEIGHT, MASK_WIDTH, MASK_HEIGHT);
    basic_2d_dev(start, mask, result, WIDTH, HEIGHT, MASK_WIDTH, MASK_HEIGHT);
    gettimeofday(&end, NULL);
    print_image(result, WIDTH, HEIGHT);    
    print_time(begin, end);
    print_divider();

/*    
    // loop thru the same mask on the result 
    while (i < LIMIT) {
        basic_2d_host(start, mask, result, WIDTH, HEIGHT, MASK_WIDTH, MASK_HEIGHT);
        normalize_image(result, WIDTH, HEIGHT, RANGE);
        print_image(result, WIDTH, HEIGHT);
        temp = result;
        result = start;
        start = temp;
        ++i;
    }
*/

    // free memory
    free(start);
    free(result);
//    free(mask);

    return 0;
}





