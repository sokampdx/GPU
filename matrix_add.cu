#include "hip/hip_runtime.h"
/*
Kam Pui So (Anthony)
CS510 GPU
Project Group A

Appliction:
Matrix Addition base on CUDA TOOLKIT Documentation

*/


#include <sys/time.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//global
const int TESTSIZE[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024};
const int MAX_TEST = 11;
const float MAX_FLOAT = 100.0f;
const int REPEAT = 10000;
const int DIMX = 94586;
const int DIMY = 75684;


// row major matrix struct
typedef struct {
	int width;
	int height;
	float* elements;
} matrix;

typedef struct{
	int x;
	int y;
} blocksize;


// print divider
void printDivider() {
	printf("-------------------------------\n");
}

// create randomize matrix
void createRandomMatrix(matrix sourceMatrix) {
	int height = sourceMatrix.height;
	int width = sourceMatrix.width;

	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
//		(*sourceMatrix).elements[(y * width) + x] = ((float) x+y) * 0.1;
			sourceMatrix.elements[(y * width) + x] = (float) rand() / (float) (RAND_MAX/MAX_FLOAT);
		}
	}
}


// print matrix
void printMatrix(const matrix valueMatrix) {
	int height = valueMatrix.height;
	int width = valueMatrix.width;

	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			printf("%.2f ", valueMatrix.elements[(y * width) + x]);
		}
		printf("\n");
	}
	printDivider();
}


// sequential matrix addition
void addMatrix(const matrix A, const matrix B, matrix result) {
	int height = result.height;
	int width = result.width;
	int index = 0;

	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			index = y * width + x;
			result.elements[index] = A.elements[index] + B.elements[index];
		}
	}
}


// print error code
void printError(char *message, hipError_t error) {
	char errorString[255];
	strcpy(errorString, hipGetErrorString(error));
	if (strcmp(errorString, "no error") == 1)
		printf("%s: %s\n", message, hipGetErrorString(error));
}


// Kernel code - matrix addition
// A + B = C
__global__ void matrixAddKernel(const matrix A, const matrix B, matrix C) {
	int height = C.height;
	int width = C.width;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// check if row & col are within matrix size
	if ((row > height) || (col > width)) return;

	int index = row * width + col;

	C.elements[index] = A.elements[index] + B.elements[index]; 
}


// Host code - matrix addition
// A + B = C
// block size is determine at runtime
void matrixAddHost(const matrix A, const matrix B, matrix C, const blocksize dimension) {
	// variable declaration
	matrix A_device, B_device, C_device;
	hipError_t err;
	int height = C.height;
	int width = C.width;
	size_t size = height * width * sizeof(float);

	A_device.width = B_device.width = C_device.width = width;
	A_device.height = B_device.height = C_device.height = height;

	// load A and B to device memory
	err = hipMalloc(&A_device.elements, size);
	printError("CUDA malloc A", err);
	err = hipMemcpy(A_device.elements, A.elements, size, hipMemcpyHostToDevice);
	printError("Copy A to device", err);

	err = hipMalloc(&B_device.elements, size);
	printError("CUDA malloc B", err);
	err = hipMemcpy(B_device.elements, B.elements, size, hipMemcpyHostToDevice);
	printError("Copy B to device", err);

	// allocate C in device memory
	err = hipMalloc(&C_device.elements, size);
	printError("CUDA malloc C", err);
	
	// invoke kernel
	dim3 dimBlock(dimension.x, dimension.y);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
	matrixAddKernel<<<dimGrid, dimBlock>>>(A_device, B_device, C_device);
	err = hipDeviceSynchronize();
	printError("Run kernel", err);

	// read C back from device memory
	err = hipMemcpy(C.elements, C_device.elements, size, hipMemcpyDeviceToHost);
	printError("Copy C off of device", err);

	// free device memory
	hipFree(A_device.elements);
	hipFree(B_device.elements);
	hipFree(C_device.elements);
}

// print result
void printResult(const timeval start, const timeval end, const blocksize testSize) {
	printf("Result (x y micro-second), %d, %d, %ld\n", testSize.x, testSize.y, ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec )));
}


// run sizing test on kernel
void runSizeTest(const matrix A, const matrix B, matrix C) {
	blocksize currentSize;
	int i = 0;
	int x, y;
	struct timeval start, end;

	// set up test loop
	while ( i < REPEAT) {
		x = rand() % MAX_TEST;
		y = rand() % MAX_TEST;
		currentSize.x = TESTSIZE[x];
		currentSize.y = TESTSIZE[y];

		gettimeofday(&start, NULL);
		matrixAddHost(A, B, C, currentSize);
		gettimeofday(&end, NULL);
		printResult(start, end, currentSize);
//		printMatrix(C);

		++i;
	}
}


// main function
// usage ./a.out dimensionX dimensionY
int main (int argc, char*argv[]) {
	matrix A, B, C;
//	int dimX = atoi(argv[1]);
//	int dimY = atoi(argv[2]);
	int dimX = DIMX;
	int dimY = DIMY;

	printf("dimension, %d, %d\n", dimX, dimY);

	// initialize random seed
	srand(time(NULL));

	// setup initial matrix
	A.height = dimX;
	A.width = dimY;
	A.elements = (float*) malloc(dimX * dimY * sizeof(float));

	B.height = dimX;
	B.width = dimY;
	B.elements = (float*) malloc(dimX * dimY * sizeof(float));

	C.height = dimX;
	C.width = dimY;
	C.elements = (float*) malloc(dimX * dimY * sizeof(float));

	// create random matrix
	createRandomMatrix(A);
	createRandomMatrix(B);

	// print initial matrix
//	printMatrix(A);
//	printMatrix(B);

	// tranditional addition
//	addMatrix(A, B, C);

	// CUDA addition
	runSizeTest(A, B, C);

	// free matrix
	free(A.elements);
	free(B.elements);
	free(C.elements);

	return 0;
}















