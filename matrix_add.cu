#include "hip/hip_runtime.h"
/*
Kam Pui So (Anthony)
CS510 GPU
Project Group A

Appliction:
Matrix Addition base on CUDA TOOLKIT Documentation

*/


#include <sys/time.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//global
const int TESTSIZE[] = {1, 5, 7, 11, 13, 16, 23, 29, 32, 47, 64};
const int MAX_TEST = 11;
const float MAX_FLOAT = 100.0f;
const int REPEAT = 10;

// row major matrix struct
typedef struct {
	int width;
	int height;
	float* elements;
} matrix;

typedef struct{
	int x;
	int y;
} blocksize;


// print divider
void printDivider() {
	printf("-------------------------------\n");
}

// create randomize matrix
void createRandomMatrix(matrix sourceMatrix) {
	int height = sourceMatrix.height;
	int width = sourceMatrix.width;

	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
//		(*sourceMatrix).elements[(y * width) + x] = ((float) x+y) * 0.1;
			sourceMatrix.elements[(y * width) + x] = (float) rand() / (float) (RAND_MAX/MAX_FLOAT);
		}
	}
}


// print matrix
void printMatrix(const matrix valueMatrix) {
	int height = valueMatrix.height;
	int width = valueMatrix.width;

	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			printf("%.2f ", valueMatrix.elements[(y * width) + x]);
		}
		printf("\n");
	}
	printDivider();
}


// sequential matrix addition
void addMatrix(const matrix A, const matrix B, matrix result) {
	int height = result.height;
	int width = result.width;
	int index = 0;

	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			index = y * width + x;
			result.elements[index] = A.elements[index] + B.elements[index];
		}
	}
}


// print error code
void printError(char *message, hipError_t error) {
	char errorString[255];
	strcpy(errorString, hipGetErrorString(error));
	if (strcmp(errorString, "no error") == 1)
		printf("%s: %s\n", message, hipGetErrorString(error));
}


// Kernel code - matrix addition
// A + B = C
__global__ void matrixAddKernel(const matrix A, const matrix B, matrix C) {
	int height = C.height;
	int width = C.width;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// check if row & col are within matrix size
	if ((row > height) || (col > width)) return;

	int index = row * width + col;

	C.elements[index] = A.elements[index] + B.elements[index]; 
}


// Host code - matrix addition
// A + B = C
// block size is determine at runtime
void matrixAddHost(const matrix A, const matrix B, matrix C, const blocksize dimension) {
	// variable declaration
	matrix A_device, B_device, C_device;
	hipError_t err;
	int height = C.height;
	int width = C.width;
	size_t size = height * width * sizeof(float);

	A_device.width = B_device.width = C_device.width = width;
	A_device.height = B_device.height = C_device.height = height;

	// load A and B to device memory
	err = hipMalloc(&A_device.elements, size);
	printError("CUDA malloc A", err);
	err = hipMemcpy(A_device.elements, A.elements, size, hipMemcpyHostToDevice);
	printError("Copy A to device", err);

	err = hipMalloc(&B_device.elements, size);
	printError("CUDA malloc B", err);
	err = hipMemcpy(B_device.elements, B.elements, size, hipMemcpyHostToDevice);
	printError("Copy B to device", err);

	// allocate C in device memory
	err = hipMalloc(&C_device.elements, size);
	printError("CUDA malloc C", err);
	
	// invoke kernel
	dim3 dimBlock(dimension.x, dimension.y);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
	matrixAddKernel<<<dimGrid, dimBlock>>>(A_device, B_device, C_device);
	err = hipDeviceSynchronize();
	printError("Run kernel", err);

	// read C back from device memory
	err = hipMemcpy(C.elements, C_device.elements, size, hipMemcpyDeviceToHost);
	printError("Copy C off of device", err);

	// free device memory
	hipFree(A_device.elements);
	hipFree(B_device.elements);
	hipFree(C_device.elements);
}





// main function
// usage ./a.out dimensionX dimensionY
int main (int argc, char*argv[]) {
	matrix A, B, C;
	blocksize currentSize;
	int x, y;
	int dimX = atoi(argv[1]);
	int dimY = atoi(argv[2]);

	// initialize random seed
	srand(time(NULL));



	// setup initial matrix
	A.height = dimX;
	A.width = dimY;
	A.elements = (float*) malloc(dimX * dimY * sizeof(float));

	B.height = dimX;
	B.width = dimY;
	B.elements = (float*) malloc(dimX * dimY * sizeof(float));

	C.height = dimX;
	C.width = dimY;
	C.elements = (float*) malloc(dimX * dimY * sizeof(float));

	// create random matrix
	createRandomMatrix(A);
	createRandomMatrix(B);

	// print initial matrix
	printMatrix(A);
	printMatrix(B);


/*
	// tranditional addition
	addMatrix(A, B, C);
*/

	// CUDA addition
	x = rand() % MAX_TEST;
	y = rand() % MAX_TEST;
	currentSize.x = TESTSIZE[x];
	currentSize.y = TESTSIZE[y];

	printf("x=%d, y=%d\n", x, y);

	matrixAddHost(A, B, C, currentSize);


	// print result
	printMatrix(C);


	// free matrix
	free(A.elements);
	free(B.elements);
	free(C.elements);

	return 0;
}














